#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cfloat>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "my-cuda-function-kernel-ansi.h"
#include <algorithm>

#define CUDA_CALL(ret) \
{\
  if((ret) != hipSuccess) { \
  printf("Error at %s:%d\n", __FILE__, __LINE__); \
  printf("Error code %s", hipGetErrorString(ret)); \
  exit(-1); \
  } \
  hipDeviceSynchronize(); \
}

template<typename Real>
__device__
static void __insert_sort(Real *__first, Real *__last)
{
	if (__first == __last)
		return;
	Real *p;
	for (Real *iter = __first + 1; iter != __last; ++iter)
	{
		Real tmp = *iter;
		for (p = iter; p != __first && tmp < *(p - 1); --p)
			*p = *(p - 1);
		*p = tmp;
	}
}

template<typename Real>
__device__
static Real* __partition(Real *__first, Real *__last, Real __pivot)
{
	while(true)
	{
		while (*__first < __pivot)
			++__first;
		--__last;
		while (__pivot < *__last)
			--__last;
		if(!(__first < __last))
			return __first;
		//swap two number, use plus function to swap two number will lost precision.
		{
			Real temp = *__first;
			*__first = *__last;
			*__last = temp;
		}
		++__first;
	}
}

template<typename Real>
__device__
static void _partition(Real *__first, Real *__nth, Real *__last)
{
	while(__last - __first > 3)
	{
		Real *__cut = __partition(__first, __last, *(__first + (__last - __first) / 2));
		if (__cut <= __nth)
			__first = __cut;
		else
			__last = __cut;
	}
	__insert_sort(__first, __last);
}

template<typename Real>
__global__
static void _gmm_select(Real *data, MatrixDim d, Real *copydata, MatrixDim c_d, int32_cuda num_gselect, int32_cuda *gmm_selected)
{
	int32_cuda row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row < d.rows)
	{
		_partition(copydata + row * c_d.stride, copydata + row * c_d.stride + c_d.cols - num_gselect, copydata + row * c_d.stride + c_d.cols);
		Real thresh = copydata[row * c_d.stride + c_d.cols - num_gselect];
		int32_cuda index = 0;
		for (int32_cuda j = 0; j < d.cols; j++)
			if (*(data + row * d.stride + j) >= thresh)
			{
				if(index < 20)
				{
					*(gmm_selected + row * num_gselect + index) = j;
					++index;
				}
				else break;
			}
		__syncthreads();
	}
}

template<typename Real>
__global__
static void printdata(Real *data, MatrixDim d)
{
	for (int r = 0; r < 2; r++)
	{
		printf("[");
		for (int c = 0; c < d.cols; c++)
			printf(" %g", data[1000 * d.stride + c]);
		printf(" ]");
	}
}

template<typename Real>
__host__
static void _my_cuda_gmm_select(int32_cuda Gr, int32_cuda Bl, Real *data, MatrixDim d, int32_cuda num_gselect, int32_cuda *gmm_out)
{
	int32_cuda *selected_gauss;
	Real *copydata;
	size_t pitch;
	MatrixDim c_d;
	c_d.rows = d.rows;
	c_d.cols = d.cols;
	CUDA_CALL(hipMallocPitch((void **)&copydata, &pitch, d.cols * sizeof(Real), d.rows));
	c_d.stride = pitch / sizeof(Real);
	CUDA_CALL(hipMemcpy2D(copydata, c_d.stride * sizeof(Real), data, d.stride * sizeof(Real), d.cols * sizeof(Real), d.rows, hipMemcpyDeviceToDevice));
	CUDA_CALL(hipMalloc((void **)&selected_gauss, d.rows * num_gselect * sizeof(int32_cuda)));
	hipDeviceSynchronize();
//	printdata<<<1,1>>>(copydata, c_d);
	_gmm_select<<<Gr, Bl>>>(data, d, copydata, c_d, num_gselect, selected_gauss);
	CUDA_CALL(hipMemcpy(gmm_out, selected_gauss, d.rows * num_gselect * sizeof(int32_cuda), hipMemcpyDeviceToHost));
	CUDA_CALL(hipFree(selected_gauss));
	CUDA_CALL(hipFree(copydata));
}


template<typename Real>
__host__
static void _my_cuda_compute_fft(Real *data, int32_cuda dim)
{
	hipfftComplex *CompData = (hipfftComplex *)malloc(dim * sizeof(hipfftComplex));
	for (int32_cuda i = 0; i < dim; i++)
	{
		CompData[i].x = data[i];
		CompData[i].y = 0;
	}
	hipfftComplex *devData;
	CUDA_CALL(hipMalloc((void **)&devData, dim * sizeof(hipfftComplex)));
	CUDA_CALL(hipMemcpy(devData, CompData, dim * sizeof(CompData), hipMemcpyHostToDevice));

	hipfftHandle plan;
	hipfftPlan1d(&plan, dim, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, devData, devData, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	CUDA_CALL(hipMemcpy(CompData, devData, dim * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
	for (int32_cuda i = 0; i < dim / 2; i++)
	{
		data[2 * i] = CompData[i].x;
		data[2 * i + 1] = CompData[i].y;
	}
	data[1] = CompData[dim / 2].x;
	CUDA_CALL(hipFree(devData));
	free(CompData);
}


//define in my-cuda-function-kernel-ansi.h
void _F_my_cuda_compute_fft(float *data, int32_cuda dim)
{
	_my_cuda_compute_fft(data, dim);
}

void _D_my_cuda_compute_fft(double *data, int32_cuda dim)
{
	_my_cuda_compute_fft(data, dim);
}

void _F_my_cuda_gmm_select(int32_cuda Gr, int32_cuda Bl, float *data, MatrixDim d, int32_cuda num_gselect, int32_cuda *gmm_out)
{
	_my_cuda_gmm_select(Gr, Bl, data, d, num_gselect, gmm_out);
}

void _D_my_cuda_gmm_select(int32_cuda Gr, int32_cuda Bl, double *data, MatrixDim d, int32_cuda num_gselect, int32_cuda *gmm_out)
{
	_my_cuda_gmm_select(Gr, Bl, data, d, num_gselect, gmm_out);
}

