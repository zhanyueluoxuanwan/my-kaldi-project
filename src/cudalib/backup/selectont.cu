#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "gettime.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
     printf("Error at %s:%d\n",__FILE__,__LINE__); \
     exit(-1);}hipDeviceSynchronize();} while(0)

void test() {
	printf("\nfor a test");
}

template<typename Real>
__device__
static void __insert_sort(Real *__first, Real *__last)
{
	if (__first == __last)
		return;
	Real *p;
	for (Real *iter = __first + 1; iter != __last; ++iter)
	{
		Real tmp = *iter;
		for (p = iter; p != __first && tmp < *(p - 1); --p)
			*p = *(p - 1);
		*p = tmp;
	}
}

template<typename Real>
__device__
static Real* __partition(Real *__first, Real *__last, Real __pivot)
{
	while(true)
	{
		while (*__first < __pivot)
			++__first;
		--__last;
		while (__pivot < *__last)
			--__last;
		if(!(__first < __last))
			return __first;
		//swap two number
		{
			*__first += *__last;
			*__last = *__first - *__last;
			*__first -= *__last;
		}
		++__first;
	}
}

template<typename Real>
__device__
static void _partition(Real *__first, Real *__nth, Real *__last)
{
	while(__last - __first > 3)
	{
		Real *__cut = __partition(__first, __last, *(__first + (__last - __first) / 2));
		if (__cut <= __nth)
			__first = __cut;
		else
			__last = __cut;
	}
	__insert_sort(__first, __last);
}

template<typename Real>
__global__
static void _gmm_select(Real *data, int rows, int cols, int num_ceps, int *gmm_selected)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row < rows)
	{
		Real *dataCopy = (Real *)malloc(cols * sizeof(Real));
		//copy data
		for(int n = 0; n < cols; n++)
			dataCopy[n] = data[row * cols + n];
		//sort copy data to get greater 20 numbers;
		_partition(dataCopy, dataCopy + cols - num_ceps, dataCopy + cols);
		Real thresh = dataCopy[cols - num_ceps];
		printf("thread %d thresh is %f\toffset is %d\n", row, thresh, cols - num_ceps);
		for (int j = 0; j < cols; j++)
			if(*(data + row * cols + j) >= thresh)
			{
				*(gmm_selected + row * num_ceps) = j;
				gmm_selected++;
			}
	}
}


template<typename Real>
__host__
static int *_my_cuda_gmm_select(Real *data, int rows, int cols, int num_ceps)
{
	int threadsPerBlock = 256;
	int blockPerGrid = (rows + threadsPerBlock - 1) / threadsPerBlock;
	int *selected_gauss;
	int *host_selected_gauss = (int *)malloc(rows * num_ceps * sizeof(int));
	Real *devdata;
	CUDA_CALL(hipMalloc((void **)&selected_gauss, rows * num_ceps * sizeof(int)));
	CUDA_CALL(hipMalloc((void **)&devdata, rows * cols * sizeof(float)));
	my_time t;
	t.start();
	CUDA_CALL(hipMemcpy(devdata, data, rows * cols * sizeof(float), hipMemcpyHostToDevice));
	_gmm_select<<<blockPerGrid, threadsPerBlock>>>(devdata, rows, cols, num_ceps, selected_gauss);
//	_gmm_select<<<1, 1>>>(devdata, dim, num_ceps, selected_gauss);
	CUDA_CALL(hipMemcpy(host_selected_gauss, selected_gauss, rows * num_ceps * sizeof(int), hipMemcpyDeviceToHost));
	t.end();
	printf("gpu gmm select used time is:%lld", t.used_time());
	CUDA_CALL(hipFree(selected_gauss));
	CUDA_CALL(hipFree(devdata));
	return host_selected_gauss;
}

int main()
{
	float data[4096];
	for (int i = 0; i < 4096; i++)
		data[i] = static_cast<float>(i);
	int num_gselect = 20;
	int *p = _my_cuda_gmm_select(data, 64, 64, num_gselect);
	printf("\n");
	for (int j = 0; j < 64; j++)
	{
		for (int i = 0;i < num_gselect; i++)
			printf("p[%d]=%d  ", i, p[i]);
		printf("\n");
		p += num_gselect;
	}
	return 0;
}


